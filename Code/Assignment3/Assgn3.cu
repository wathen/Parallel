#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "time_it.h"
#include <iostream>
using namespace std;

#define N (512*512)
#define THREADS_PER_BLOCK 128

__global__ void logistic_cuda(unsigned int n, unsigned int m, float a, float *x, float *z) {
  unsigned int myId = blockDim.x*blockIdx.x + threadIdx.x;
  if(myId < n){
    for (int i = 1; i < m; ++i) {
      z[myId] = a*x[myId]*(1.0f - x[myId]);
      x[myId] = z[myId];
    }
  }
}

void logistic_ref(unsigned int n, unsigned int m, float a, float *x, float *z) {
  for (int j = 1; j < m; ++j) {
    for(int i = 1; i < n; ++i){
     z[i] = a*x[i]*(1.0f - x[i]);
     x[i] = z[i];
    }
  }

}
struct kernel_arg {
    float *x;
    unsigned int n;
};

__device__ void reduce_sum_dev(unsigned int n, float *x) {
  unsigned int myId = threadIdx.x;
  for(unsigned int m = n >> 1; m > 0; m = n >> 1) {
    n -= m;
    __syncthreads();
    if(myId < m)
      x[myId] += x[myId+n];
  }
}

__global__ void reduce_sum(unsigned int n, float *x) {
  reduce_sum_dev(n, x);
}

float reduce_sum_ref(unsigned int n, float *x) {
  float sum = 0.0;
  for(int i = 0; i < n; i++)
    sum += x[i];
  return(sum);
}

/******************************
*  dotprod: just like it sounds
*    Simple version: we only handle one block of threads
*******************************/

__global__ void dotprod(unsigned int n, float *x, float *z) {
  unsigned int blockBase = blockDim.x * blockIdx.x;
  unsigned int myId = blockBase + threadIdx.x;
  unsigned int m = min(blockDim.x, n - blockBase);

  if(myId < n)
    x[myId] *= x[myId];
  reduce_sum_dev(m, &(x[blockBase]));
  if((myId < n) && (threadIdx.x == 0))
    z[blockIdx.x] = x[myId];
}

float norm_ref(float *x, unsigned int n) {
  float sum = 0.0;
  for(int i = 0; i < n; i++)
    sum += x[i] * x[i];
  return(sqrt(sum));
}

/*****************************************************
*  print_vec: print the first few elements of a vector
******************************************************/

void print_vec(float *x, unsigned int n, const char *fmt, const char *who) {
  printf("%s = ", who);
  for(int i = 0; i < n; i++) {
    if(i > 0) printf(", ");
    printf(fmt, x[i]);
  }
  if(n > 10) printf(", ...");
  printf("\n");
}

void logistic(float *x, unsigned int a, unsigned int n, unsigned int m, float *z) {

  float *dev_x, *dev_z;
  int size = n*sizeof(float);
  hipMalloc((void**)(&dev_x), size);
  hipMalloc((void**)(&dev_z), size);
  hipMemcpy(dev_x, x, size, hipMemcpyHostToDevice);

  logistic_cuda<<<N/THREADS_PER_BLOCK , THREADS_PER_BLOCK>>>(n, m, a, dev_x, dev_z);
  hipMemcpy(z, dev_z, sizeof(size), hipMemcpyDeviceToHost);
}

float norm(float * x, unsigned int n) {

  float *z;
  float *dev_x, *dev_z;
  int size = n*sizeof(float);
  z = (float *) malloc(size);

  hipMalloc((void**)(&dev_x), size);
  hipMalloc((void**)(&dev_z), size);
  hipMemcpy(dev_x, x, size, hipMemcpyHostToDevice);

  dotprod<<<N/THREADS_PER_BLOCK , THREADS_PER_BLOCK>>>(n, dev_x, dev_z);
  reduce_sum<<<1,N/THREADS_PER_BLOCK>>>(N/THREADS_PER_BLOCK, dev_z);
  hipMemcpy(z, dev_z, sizeof(float), hipMemcpyDeviceToHost);
  return sqrt(z[0]);
}

void do_timing(void *void_arg) {
  struct kernel_arg *argk = (struct kernel_arg *)(void_arg);
  norm(argk->x, argk->n);
  hipDeviceSynchronize();
}

int main(int argc, char **argv) {
  unsigned int n = N;
  unsigned int m = 10;
  float *x, *z, *z_ref;
  float a;
  hipDeviceProp_t prop;
  struct kernel_arg argk;
  struct time_it_raw *tr = time_it_create(10);
  struct time_it_stats stats;

  int size = n*sizeof(float);
  x  = (float *)malloc(size);
  z  = (float *)malloc(size);
  z_ref = (float *)malloc(size);

  for(int i = 0; i < n; i++) {
    x[i] = i;
  }

  printf("The GPU is a %s\n", prop.name);
  printf("Cuda capability %d.%d.\n", prop.major, prop.minor);
  float p_norm = norm(x, n);
  z_ref[0] = norm_ref(x, n);


  printf("Parallel = %f, Sequential = %f\n\n", p_norm, z_ref[0]);
  argk.n = N;
  argk.x = x;
  time_it_run(tr, do_timing, (void *)(&argk));
  time_it_get_stats(tr, &stats);
  printf("mean(T) = %10.3e, std(T) = %10.3e\n", stats.mean, stats.std);

  a = 3.0;
  float *L;
  L  = (float*)malloc(size);
  logistic(x, a, n, m, L);
  logistic_ref(n, m, a, x, z);

  print_vec(z, min(10, N), "%5.3f", "z");
  print_vec(L, min(10, N), "%5.3f", "z");

  free(x);
  free(z_ref);
  exit(0);
}
