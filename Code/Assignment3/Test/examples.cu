#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#define SAXPY 1
#define VECSUM 2
#define REDUCE_SUM 3
#define DOTPROD 4
#define DEFAULT_TEST REDUCE_SUM

/************************
*  saxpy
************************/

__global__ void saxpy(unsigned int n, float a, float *x, float *y) {
  unsigned int myId = blockDim.x*blockIdx.x + threadIdx.x; // nvcc built-ins
  if(myId < n)
    y[myId] = a*x[myId] + y[myId];
  }

void saxpy_ref(unsigned int n, float a, float *x, float *y, float *z) {
  for(int i = 0; i < n; i++)
    z[i] = a*x[i] + y[i];
}


/***********************************************************
*  vecadd: really just a simple case of saxpy, with a = 1
*          to be non-conformists, we write the sum into a
*          result vector rather than computing it in-place
************************************************************/

__global__ void vecadd(unsigned int n, float *x, float *y, float *z) {
  unsigned int myId = blockDim.x*blockIdx.x + threadIdx.x;
  if(myId < n)
    z[myId] = x[myId] + y[myId];
}

void vecadd_ref(unsigned int n, float *x, float *y, float *z) {
  for(int i = 0; i < n; i++)
    z[i] = x[i] + y[i];
}

/**************************************************************
 *  reduce_sum: compute the sum of the elements of an array
 *    Simple version: we only handle one block of threads
 ***************************************************************/

__device__ void reduce_sum_dev(unsigned int n, float *x) {
  unsigned int myId = threadIdx.x;
  for(unsigned int m = n >> 1; m > 0; m = n >> 1) {
    n -= m;
    __syncthreads();
    if(myId < m)
      x[myId] += x[myId+n];
  }
}

__global__ void reduce_sum(unsigned int n, float *x) {
  reduce_sum_dev(n, x);
}

float reduce_sum_ref(unsigned int n, float *x) {
  float sum = 0.0;
  for(int i = 0; i < n; i++)
    sum += x[i];
  return(sum);
}

/******************************
*  dotprod: just like it sounds
*    Simple version: we only handle one block of threads
*******************************/

__global__ void dotprod(unsigned int n, float *x, float *y, float *z) {
  unsigned int blockBase = blockDim.x * blockIdx.x;
  unsigned int myId = blockBase + threadIdx.x;
  unsigned int m = min(blockDim.x, n - blockBase);

  if(myId < n)
    x[myId] *= y[myId];
  reduce_sum_dev(m, &(x[blockBase]));
  if((myId < n) && (threadIdx.x == 0))
    z[blockIdx.x] = x[myId];
}

float dotprod_ref(unsigned int n, float *x, float *y) {
  float sum = 0.0;
  for(int i = 0; i < n; i++)
    sum += x[i] * y[i];
  return(sum);
}

/********************************************************************
 *  dotprod2: just like it sounds
 *    First phase of a dotprod.  Each block computes its part of the
 *    dot-product and stores the result in the z array.  A subsequent
 *    launch of the reduce_sum kernel completes the calculation.
 ********************************************************************/

/*****************************************************
*  print_vec: print the first few elements of a vector
******************************************************/

void print_vec(float *x, unsigned int n, const char *fmt, const char *who) {
  printf("%s = ", who);
  for(int i = 0; i < n; i++) {
    if(i > 0) printf(", ");
    printf(fmt, x[i]);
  }
  if(n > 10) printf(", ...");
  printf("\n");
}

/*****************************************************
*  near(x, y): true if x and y are "nearly" equal
******************************************************/
int near(unsigned int n, float x, float y) {
  return(abs(x-y) < max(10.0, sqrt((float)n))*1.0e-7*max(1.0, max(abs(x), abs(y))));
}

int main(int argc, char **argv) {
  unsigned int n = (argc >= 2) ? atoi(argv[1]) : 1000000;
  unsigned int nn = n;
  unsigned int what = (argc >= 3) ? atoi(argv[2]) : DEFAULT_TEST;
  float *x, *y, *z, *z_ref;
  float *dev_x, *dev_y, *dev_z;
  float a;
  hipDeviceProp_t prop;

  int ndev;
  hipGetDeviceCount(&ndev);
  // if(ndev < 1) {
  //   fprintf(stderr, "No CUDA device found\n");
  //   exit(-1);
  // }
  hipGetDeviceProperties(&prop, 0);

  int size = n*sizeof(float);
  x  = (float *)malloc(size);
  y  = (float *)malloc(size);
  z  = (float *)malloc(size);
  z_ref = (float *)malloc(size);

  // Use a logistic map to make some pseudo-random numbers
  // It's fast, but the distribution isn't very uniform, and
  //   the other statistical properties are lousy.  But it's
  //   fast, and that's all we need for some simple tests.
  x[0] = 0.123;
  y[0] = sqrt(0.3);
  for(int i = 1; i < n; i++) {
    x[i] = 3.8*x[i-1]*(1.0 - x[i-1]);
    y[i] = 3.9*y[i-1]*(1.0 - y[i-1]);
  }

  printf("The GPU is a %s\n", prop.name);
  printf("Cuda capability %d.%d.\n", prop.major, prop.minor);
  print_vec(x, min(10, n), "%5.3f", "x");
  print_vec(y, min(10, n), "%5.3f", "y");

  hipMalloc((void**)(&dev_x), size);
  hipMalloc((void**)(&dev_y), size);
  hipMalloc((void**)(&dev_z), size);
  hipMemcpy(dev_x, x, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_y, y, size, hipMemcpyHostToDevice);

  switch(what) {
    case SAXPY:
      a = 3.0;
      saxpy<<<ceil(n/256.0),256>>>(n, a, dev_x, dev_y);
      printf("a: size = %d, z=%016llx, dev_y=%016llx\n", size, z, dev_y);
      hipMemcpy(z, dev_y, size, hipMemcpyDeviceToHost);
      printf("b\n");
      saxpy_ref(n, a, x, y, z_ref);
      break;
    case VECSUM:
      vecadd<<<ceil(n/256.0),256>>>(n, dev_x, dev_y, dev_z);
      hipMemcpy(z, dev_z, size, hipMemcpyDeviceToHost);
      vecadd_ref(n, x, y, z_ref);
      break;
    case REDUCE_SUM:
      // __synchronize() only works within a single thread block.
      if(n > prop.maxThreadsPerBlock) {
        fprintf(stderr, "reduce_sum: array size too big, max = %d\n",
                prop.maxThreadsPerBlock);
        exit(-1);
      }
      reduce_sum<<<1,n>>>(n, dev_x);
      hipMemcpy(z, dev_x, sizeof(float), hipMemcpyDeviceToHost);
      z_ref[0] = reduce_sum_ref(n, x);
      nn = 1;
      break;
    case DOTPROD:
      unsigned int blksize;
      unsigned int nblks;
      blksize = prop.maxThreadsPerBlock;
      nblks = ceil(((float)(n))/((float)(blksize)));
      if(nblks > blksize) {
        fprintf(stderr, "dotprod: array size too big, max = %d\n",
                blksize*blksize);
        exit(-1);
      }
      dotprod<<<nblks,blksize>>>(n, dev_x, dev_y, dev_z);
      reduce_sum<<<1,nblks>>>(nblks, dev_z);
      hipMemcpy(z, dev_z, sizeof(float), hipMemcpyDeviceToHost);
      z_ref[0] = dotprod_ref(n, x, y);
      nn = 1;
      break;
    default:
      fprintf(stderr, "ERROR: unknown test case -- %d\n", what);
      exit(-1);
  }

  for(int i = 0; i < nn; i++) { // check the result
    if(!near(n, z[i], z_ref[i])) {
      fprintf(stderr, "ERROR: i=%d: z[i] = %15.10f, z_ref[i] = %15.10f\n", z[i], z_ref[i]);
      exit(-1);
    }
  }
  print_vec(z, min(10, nn), "%5.3f", "z");
  printf("The results match!\n");

  hipFree(dev_x);
  hipFree(dev_y);
  hipFree(dev_z);
  free(x);
  free(y);
  free(z);
  free(z_ref);
  exit(0);
}
